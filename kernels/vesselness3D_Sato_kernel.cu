
#include <hip/hip_runtime.h>

__device__ float phi(float eig1, float eig2, float gamma) {
	if (eig1 < 0.f)
		return __powf(eig1/eig2, gamma);

	return 0.f;
}

__device__ float omega(float eig1, float eig2, float gamma, float alpha) {
	eig2 = abs(eig2);

	if (eig1 <= 0.f)
		return __powf(1.f + eig1/eig2, gamma);

	if (eig1 < eig2/alpha)
		return __powf(1.f - alpha*eig1/eig2, gamma);

	return 0.f;
}


// Vesselness device kernel
__global__ void vesselness3DKernel(
	const int num_elements,
	const float * eig1,
	const float * eig2,
	const float * eig3,
	float * V,
	float gamma,
	float alpha
){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= num_elements) return;

	if (eig2[i] < 0)
		V[i] = abs(eig3[i]) * phi(eig2[i],eig3[i],gamma) * omega(eig1[i],eig2[i],gamma,alpha);
	else
		V[i] = 0;
}
